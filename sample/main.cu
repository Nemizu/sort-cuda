#include "../header/SortCuda.h"
#include <chrono>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#define TIMES 100
#define THREAD 100

int main(int argc, char **argv) {
    int elements = 10000;//number of arry elements
	int thread = 128;
	if(argc > 1){
		elements = atoi(argv[1]);
	}
	if(argc > 2){
		thread = atoi(argv[2]);
	}
	size_t size = elements *sizeof(int);
	float time = 0.0;
	int *h_ary, *d_ary,*d_resary;

	/*malloc of houst array*/
	if((h_ary = (int *)malloc(size)) == NULL) {
		fprintf(stderr, "Out of memory, exit.\n");
		exit(1);
	}

	/*init host array*/
	init_array(h_ary,elements,1);
	shuffle_array(h_ary,elements);

	/*malloc of device array*/	
	hipMalloc((void **)&d_ary, size);
	hipMalloc((void **)&d_resary, size);

	/*init device array*/
	hipMemcpy(d_ary, h_ary, size, hipMemcpyHostToDevice);
	hipMemcpy(d_resary, h_ary, size, hipMemcpyHostToDevice);

	/*precheck*/
	testsort<<<elements/thread+1,thread>>>(d_ary,d_resary,elements);
	hipMemcpy(h_ary,d_resary, size, hipMemcpyDeviceToHost);
	if(!check_array(h_ary,elements))exit(1);
	hipMemcpy(h_ary,d_ary, size, hipMemcpyDeviceToHost);
	
	auto start = std::chrono::system_clock::now();
	for(int loop = 0; loop < TIMES; loop++){
		testsort<<<elements/thread+1,thread>>>(d_ary,d_resary,elements);
		hipDeviceSynchronize();
	}
	auto end = std::chrono::system_clock::now();
	time = (std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count());
	
	free(h_ary);
	hipFree(d_ary);
	hipFree(d_resary);

	printf("%f(ms)\n",time/TIMES/1000000);
	
	return 0;
}


