
#include <hip/hip_runtime.h>
__global__ void bitonicSort_Dev(int* ary,int i,int j){
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    int ip = id^i;

    if(ip>id){
        if((id&j)==0){
            if(ary[id]>ary[ip]){
                int tmp = ary[id];
                ary[id] = ary[ip];
                ary[ip] = tmp;
            }
        }
        if((id&j)!=0){
            if(ary[id]<ary[ip]){
                int tmp = ary[id];
                ary[id] = ary[ip];
                ary[ip] = tmp;
            }
        }
    }
}

void bitonicsort(int* d_ary,int elements,int block,int thread){
    
    for(int i = 2; i <= elements; elements *= 2){
        for (int j = i / 2; j > 0; j /= 2){
            bitonicSort_Dev<<<block,thread>>>(d_ary,i,j);
            hipDeviceSynchronize();
        }
    }

}

__global__ void testsort(int* ary,int* resary,int elements){
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if(id < elements){
        int number = 0;
        for(int i = 1;i<elements;i++){
            if(ary[id]>ary[i]){
                number++;
            }
        }
        resary[number] = ary[id];
    }
    
}





/*__global__ void bitonicsort(int* ary,int elements){
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    //sorting
    for(int length = 2; length <= elements; length *= 2){
        for (int mlength = length / 2; mlength > 0; mlength /= 2){
            int ixj = id ^ mlength;
            
			if (ixj > id) {
				int	tmp;
                if ((id & length) == 0) {
                    if (ary[id] > ary[ixj]) {
						tmp = ary[ixj];
						ary[ixj] = ary[id];
						ary[id] = tmp;
                    }
                } else {
                    if (ary[id] < ary[ixj]) {
                        tmp = ary[ixj];
						ary[ixj] = ary[id];
						ary[id] = tmp;
                    }
                }
            }
            __threadfence();
            __syncthreads();
        }
    }
    return;
}
*/