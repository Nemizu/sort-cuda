
#include <hip/hip_runtime.h>
#include<stdlib.h>

void init_array(int* ary,int elements,int step){
    for(int i=0;i<elements;i+=step){
        ary[i] = i; 
    }
	return;
}

void shuffle_array(int* ary,int elements){
	//rundom
    for(int i=0;i<elements;i++){
        int r = rand()%elements;
        int tmp = ary[i];
        ary[i] = ary[r];
        ary[r] = tmp; 
    }
	return;
}